#include "hip/hip_runtime.h"
#define _POSIX_C_SOURCE 200809L
// #define _XOPEN_SOURCE 700
#include <stdio.h> // I/O library
#include <stdlib.h> // posix_memalign library
#include <string.h>
#include <omp.h> // #pragma omp library
// #include <mpi.h> // mpi library
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#define TILEWIDTH 16

__global__ void jacobi_kernel(double *grid, double *grid_new);

int main()
{
    int err;
    hipError_t cuda_err;
    double *grid_h, *grid_d, *grid_new_d;
    double t0, dt, chk;
#if DUMP == 1
    char myfile[32];
#endif

    int nblocksX = (GLX % TILEWIDTH == 0) ? (GLX / TILEWIDTH) : (GLX / TILEWIDTH + 1);
    int nblocksY = (GLY % TILEWIDTH == 0) ? (GLY / TILEWIDTH) : (GLY / TILEWIDTH + 1);

    dim3 dimBlock(TILEWIDTH, TILEWIDTH, 1);
    dim3 dimGrid(nblocksX, nblocksY, 1);

    err = posix_memalign((void **) &grid_h, 4096, GX * GY * sizeof(double));
    if (err != 0)
    {
        fprintf(stderr, "grid_h posix_memaling error!\n");
        exit(-1);
    }

    cuda_err = hipMalloc((void **) &grid_d, GX * GY * sizeof(double));
    if (cuda_err != hipSuccess)
    {
        fprintf(stderr, "grid_d hipMalloc error!\n");
        exit(-1);
    }
    cuda_err = hipMalloc((void **) &grid_new_d, GX * GY * sizeof(double));
    if (cuda_err != hipSuccess)
    {
        fprintf(stderr, "grid_new_d hipMalloc error!\n");
        exit(-1);
    }

    init(grid_h);

#if DUMP == 1
    sprintf(myfile, "video/grid-%07d", 0);
    dump(grid_h, myfile);
#endif

    t0 = omp_get_wtime();

    cuda_err = hipMemcpy(grid_d, grid_h, GX * GY * sizeof(double), hipMemcpyHostToDevice);
    if (cuda_err != hipSuccess)
    {
        fprintf(stderr, "grid_d hipMemcpy HostToDevice error!\n");
        exit(-1);
    }

    cuda_err = hipMemcpy(grid_new_d, grid_d, GX * GY * sizeof(double), hipMemcpyDeviceToDevice);
    if (cuda_err != hipSuccess)
    {
        fprintf(stderr, "grid_new_d hipMemcpy DeviceToDevice error!\n");
        exit(-1);
    }

    for(int iter = 1; iter <= MAXITER; iter++)
    {
        // grid_new_d <-- grid_d
        jacobi_kernel <<<dimGrid, dimBlock>>> (grid_d, grid_new_d);

        hipDeviceSynchronize();

        iter++;

        // grid_d <-- grid_new_d
        jacobi_kernel <<<dimGrid, dimBlock>>> (grid_new_d, grid_d);

        hipDeviceSynchronize();

        cuda_err = hipMemcpy(grid_h, grid_d, GX * GY * sizeof(double), hipMemcpyDeviceToHost);
        if (cuda_err != hipSuccess)
        {
            fprintf(stderr, "grid_d hipMemcpy DeviceToHost error!\n");
            exit(-1);
        }

#if DUMP == 1
        if (iter % DUMPSTEP == 0)
        {
            sprintf(myfile, "video/grid-%07d", iter);
            dump(grid_h, myfile);
        }
#endif

    }

    dt = omp_get_wtime() - t0;

    hipFree(grid_d);
    hipFree(grid_new_d);

#if DUMP == 1
    sprintf(myfile, "video/grid-%07d", MAXITER);
    dump(grid_h, myfile);
#endif

    chk = checksum(grid_h);

    printf("[statistics] %dx%d  %d iter  dt: %.3f msec  dt/iter: %.3f usec  GFLOPS: %.3f  checksum: %f\n",
        GLX, GLY, MAXITER, dt * 1e3, dt * 1e6 / (double)MAXITER, 5.0 * (double)MAXITER * (double)GLX * (double)GLY / (dt * 1e6), chk);

    free(grid_h);

    return 0;
}

__global__ void jacobi_kernel(double *grid, double *grid_new)
{
    // __shared__ double gridS[TILEWIDTH + 2][TILEWIDTH + 2];

    int idx_x = blockIdx.x * blockDim.x + threadIdx.x + HX; // idx_x pixel
    int idx_y = blockIdx.y * blockDim.y + threadIdx.y + HY; // idx_y pixel

    if (idx_x < GX - HX && idx_y < GY - HY)
    {
        // int x = threadIdx.x + 1;
        // int y = threadIdx.y + 1;
        
        // if (y == 1)
        // {
        //     gridS[y - 1][x] = grid[(idx_y - 1) * GX + idx_x];
        // }
        // else if (y == blockDim.y)
        // {
        //     gridS[y + 1][x] = grid[(idx_y + 1) * GX + idx_x];
        // }
        // if (x == 1)
        // {
        //     gridS[y][x - 1] = grid[idx_y * GX + idx_x - 1];
        // }
        // else if (x == blockDim.x)
        // {
        //     gridS[y][x + 1] = grid[idx_y * GX + idx_x + 1];
        // }
        // gridS[y][x] = grid[idx_y * GX + idx_x];
        
        // __syncthreads();

        // grid_new[idx_y * GX + idx_x] = (gridS[y][x] + gridS[y - 1][x] + gridS[y + 1][x] + gridS[y][x - 1] + gridS[y][x + 1]) / 5;

        // __syncthreads();

        grid_new[idx_y * GX + idx_x] = (grid[idx_y * GX + idx_x] + grid[(idx_y - 1) * GX + idx_x] + grid[(idx_y + 1) * GX + idx_x] + grid[idx_y * GX + idx_x - 1] + grid[idx_y * GX + idx_x + 1]) / 5;
    }
}