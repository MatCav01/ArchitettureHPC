// #define _POSIX_C_SOURCE 200809L
#include <stdio.h> // I/O library
#include <stdlib.h> // posix_memalign library
#include <omp.h> // #pragma omp library
#include <hip/hip_runtime.h> // cuda library

#define THREADS_PER_BLOCK 128

__global__ void saxpy(long L, float a, float *X, float *Y)
{
    long idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < L)
    {
        Y[idx] = a * X[idx] + Y[idx];
    }
}

int main()
{
    float *X_h, *Y_h, *X_d, *Y_d;
    float a = 17.17;
    long L = 1e8;
    double t0, dt;
    double sum = 0.0;
    int n_blocks;

    if (L % THREADS_PER_BLOCK == 0)
    {
        n_blocks = L / THREADS_PER_BLOCK;
    }
    else
    {
        n_blocks = L / THREADS_PER_BLOCK + 1;
    }

    dim3 dimBlock (THREADS_PER_BLOCK, 1, 1);
    dim3 dimGrid (n_blocks, 1, 1);

    posix_memalign((void **) &X_h, 4096, L * sizeof(float));
    posix_memalign((void **) &Y_h, 4096, L * sizeof(float));
    
    hipMalloc((void **) &X_d, L * sizeof(float));
    hipMalloc((void **) &Y_d, L * sizeof(float));

    srand48(1999);
    for (int i = 0; i < L; i++)
    {
        X_h[i] = (float) drand48();
        Y_h[i] = (float) drand48();
    }

    t0 = omp_get_wtime();

    hipMemcpy(X_d, X_h, L * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Y_d, Y_h, L * sizeof(float), hipMemcpyHostToDevice);

    saxpy <<< dimGrid, dimBlock >>> (L, a, X_d, Y_d);

    hipDeviceSynchronize();

    hipMemcpy(Y_h, Y_d, L * sizeof(float), hipMemcpyDeviceToHost);

    dt = omp_get_wtime() - t0;

    hipFree(X_d);
    hipFree(Y_d);
    
    for (int i = 0; i < L; i++)
    {
        sum += Y_h[i];
    }

    free(X_h);
    free(Y_h);

    printf("sum: %0.2f  dt: %0.2f msec\n", sum, dt * 1e3);

    return 0;
}